#include "hip/hip_runtime.h"
// -*- C++ -*-
#include <cstdio>

#include <hip/hip_runtime.h>
// These come from the cublas matrix multiplication example
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wswitch"
#include <hipblas.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#pragma GCC diagnostic pop


#include "MatrixMultiplication_cuda.cuh"

__global__
void
cudaDoNaiveMatrixMultiplication_kernel(const unsigned int matrixSize,
                                                   const double * leftMatrix,
                                                   const double * rightMatrix,
                                                   double * resultMatrix) {
  // TODO: something!
}

void
cudaDoMatrixMultiplication(const unsigned int maxNumberOfBlocks,
                           const unsigned int numberOfThreadsPerBlock,
                           const unsigned int matrixSize) {

  // TODO: something!
}


void
multiplyMatricesUsingCublas(const unsigned int matrixSize,
                            const double * leftMatrix,
                            const double * rightMatrix,
                            double * resultMatrix) {

  const unsigned int numberOfEntries = matrixSize * matrixSize;

  // allocate device memory
  double * dev_leftMatrix;
  double * dev_rightMatrix;
  double * dev_resultMatrix;
  checkCudaErrors(hipMalloc((void **) &dev_leftMatrix, numberOfEntries * sizeof(double)));
  checkCudaErrors(hipMalloc((void **) &dev_rightMatrix, numberOfEntries * sizeof(double)));
  checkCudaErrors(hipMalloc((void **) &dev_resultMatrix, numberOfEntries * sizeof(double)));
  // copy matrices to the device
  checkCudaErrors(hipMemcpy(dev_leftMatrix, leftMatrix, numberOfEntries * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dev_rightMatrix, rightMatrix, numberOfEntries * sizeof(double), hipMemcpyHostToDevice));

  const double alpha = 1.0f;
  const double beta  = 0.0f;
  hipblasHandle_t handle;

  checkCudaErrors(hipblasCreate(&handle));

  // perform the multiply
  checkCudaErrors(hipblasDgemm(handle,
                              HIPBLAS_OP_N, HIPBLAS_OP_N, // don't transpose
                              matrixSize, matrixSize, matrixSize, // sizes
                              &alpha, // no scalar premultiply
                              dev_rightMatrix, matrixSize, // left matrix
                              dev_leftMatrix, matrixSize, // right matrix
                              &beta, // don't premultiply result by anything
                              dev_resultMatrix, matrixSize));

  // copy result from device to host
  checkCudaErrors(hipMemcpy(resultMatrix, dev_resultMatrix,
                             numberOfEntries * sizeof(double),
                             hipMemcpyDeviceToHost));

  // Destroy the handle
  checkCudaErrors(hipblasDestroy(handle));

  // clean up memory
  checkCudaErrors(hipFree(dev_leftMatrix));
  checkCudaErrors(hipFree(dev_rightMatrix));
  checkCudaErrors(hipFree(dev_resultMatrix));
}
