#include "hip/hip_runtime.h"
// -*- C++ -*-
#include <cstdio>

#include <hip/hip_runtime.h>

#include "ex0_scalarIntegrator_cuda.cuh"

#define MAX_NUMBER_OF_PUPPIES 3

__constant__ double constantPuppies[MAX_NUMBER_OF_PUPPIES];

__global__
void
cudaDoScalarIntegration_kernel(double* output) {
  // block-wide reduction storage, size is determined by third kernel
  // launch argument (thing between <<< and >>>)
  extern __shared__ double contributions[];

  // TODO: do scalar integration somehow

  // reading from global memory
  *output = 5;

}

void
cudaDoScalarIntegration(const unsigned int numberOfThreadsPerBlock,
			const double startBound, double endBound,
			const double dx, double * const output) {

  // TODO: you have to do stuff in here, the junk below is just to show syntax

  // this is how to use constant memory:
  // make some stuff that we'll copy into constant memory
  double * iLikePuppies = new double[MAX_NUMBER_OF_PUPPIES];
  iLikePuppies[0] = 0.;
  iLikePuppies[1] = 1.;
  iLikePuppies[2] = 2.;
  // copy some junk into constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(constantPuppies), iLikePuppies,
                     sizeof(double) * MAX_NUMBER_OF_PUPPIES);
  delete[] iLikePuppies;


  // using global memory
  unsigned int amountOfJunk = 10;
  // make some array from which we'll copy to the device
  double * junk = new double[amountOfJunk];
  // fill it with junk
  for (int i = 0; i < amountOfJunk; ++i) {
    junk[i] = -i;
  }
  // this is going to be a pointer to memory *on the device*
  double * dev_junk;
  // allocate room on the device
  hipMalloc((void **) &dev_junk, amountOfJunk*sizeof(double));
  // copy junk from host to device
  hipMemcpy(dev_junk, junk, amountOfJunk*sizeof(double),
             hipMemcpyHostToDevice);
  delete[] junk;

  // TODO: calculate the number of blocks
  const unsigned int numberOfBlocks = 1;

  // allocate somewhere to put our result
  double *dev_output;
  hipMalloc((void **) &dev_output, 1*sizeof(double));

  // run the kernel
  cudaDoScalarIntegration_kernel<<<numberOfBlocks,
    numberOfThreadsPerBlock,
    numberOfThreadsPerBlock*sizeof(double)>>>(dev_output);

  // copy over the output
  hipMemcpy(output, dev_output, 1*sizeof(double), hipMemcpyDeviceToHost);
  // make sure that everything in flight has been completed
  hipDeviceSynchronize();

  // clean up
  hipFree(dev_junk);
  hipFree(dev_output);
}
